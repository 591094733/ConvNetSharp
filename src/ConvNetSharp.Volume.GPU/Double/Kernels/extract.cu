
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, double* __restrict x, double* __restrict output, int length, int offset) {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
        
        int batch = i / length;
        int rest = i % length;
        int batchCount = n / length;
        int width = (sizeof(x) / 2) / batchCount;

		if (i < n) {
            output[i] = x[batch * width + rest + offset];
        }
	}
}