
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, float* __restrict left, float* __restrict right, float* __restrict output, int elementPerBatch, int threshold) {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
        int batch = i / elementPerBatch;
        int rest = i % elementPerBatch;
		if (i < n) {
            if (rest < threshold) {
                output[i] = left[batch * threshold + rest];
            } else {
                output[i] = right[batch * (elementPerBatch - threshold) + rest - threshold];
            }
        }
	}
}